#include "hip/hip_runtime.h"
#include "utils/XLogger.h"
#include <cstdio>
#include <iostream>

__global__ void hi_gpu() { printf("Hello World from GPU!\n"); }

int main(int argc, char **argv) {
  XLOG_DEBUG("Hello World from CPU!");

  hi_gpu<<<1, 10>>>();
  const hipError_t err_t = hipDeviceReset();
  const std::string err_s = hipGetErrorString(err_t);
  if (err_t != hipSuccess) {
    XLOG_ERROR("CUDA error: code: {}, reason: {}", (int)err_t, err_s.c_str());
    exit(1);
  }
  return 0;
}