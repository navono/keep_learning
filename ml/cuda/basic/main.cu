
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

__global__ void hi_gpu()
{
    printf("Hello World from GPU!\n");
}

int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n");

    hi_gpu<<<1, 10>>>();

    const hipError_t err_t = hipDeviceReset();
    const std::string err_s = hipGetErrorString(err_t);
    if (err_t != hipSuccess)
    {
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);
        fprintf(stderr, "code: %d, reason: %s\n", err_t, err_s.c_str());
        exit(1);
    }
    return 0;
}