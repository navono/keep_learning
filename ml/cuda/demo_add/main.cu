#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <math.h>

#include "utils/XLogger.h"

void cpu_add(int n, float *x, float *y) {
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

void cpu_add_test() {
  int N = 1 << 25; // 30M elements

  float *x = new float[N];
  float *y = new float[N];

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  std::chrono::high_resolution_clock::time_point start =
      std::chrono::high_resolution_clock::now();

  // Run kernel on 30M elements on the CPU
  cpu_add(N, x, y);

  std::chrono::high_resolution_clock::time_point end =
      std::chrono::high_resolution_clock::now();
  auto duration =
      std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

  XLOG_INFO("add(int, float*, float*) time: {}ms", duration.count() * 1000);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  XLOG_INFO("Max error: {}", maxError);

  // Free memory
  delete[] x;
  delete[] y;
}

__global__ void add(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

void gpu_add_slow_version() {
  int N = 1 << 25;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  // 内存分配，在GPU或者CPU上统一分配内存
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  std::chrono::high_resolution_clock::time_point start =
      std::chrono::high_resolution_clock::now();

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  // execution configuration, 执行配置。按照层次从大到小可将GPU按照 grid ->
  // block -> thread划分
  // <<<numBlocks, blockSize>>>
  add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  // CPU需要等待cuda上的代码运行完毕，才能对数据进行读取
  hipDeviceSynchronize();

  std::chrono::high_resolution_clock::time_point end =
      std::chrono::high_resolution_clock::now();
  auto duration =
      std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

  XLOG_INFO("add(int, float*, float*) time: {}ms", duration.count() * 1000);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 3.0f));

  XLOG_INFO("Max error: {}", maxError);

  // Free memory
  hipFree(x);
  hipFree(y);
}

void gpu_add_normal_version() {
  int N = 1 << 25;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  // 内存分配，在GPU或者CPU上统一分配内存
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  std::chrono::high_resolution_clock::time_point start =
      std::chrono::high_resolution_clock::now();

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  // execution configuration, 执行配置。按照层次从大到小可将GPU按照 grid ->
  // block -> thread划分
  // <<<numBlocks, blockSize>>>
  add<<<32, 256>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  // CPU需要等待cuda上的代码运行完毕，才能对数据进行读取
  hipDeviceSynchronize();

  std::chrono::high_resolution_clock::time_point end =
      std::chrono::high_resolution_clock::now();
  auto duration =
      std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

  XLOG_INFO("add(int, float*, float*) time: {}ms", duration.count() * 1000);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 3.0f));

  XLOG_INFO("Max error: {}", maxError);

  // Free memory
  hipFree(x);
  hipFree(y);
}

int main(void) {
  //  cpu_add_test();
  //  gpu_add_slow_version();
  gpu_add_normal_version();

  return 0;
}