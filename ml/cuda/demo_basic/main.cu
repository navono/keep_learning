#include "hip/hip_runtime.h"
#include "utils/XLogger.h"
#include <cstdio>
#include <iostream>

// Kernel function to add the elements of two arrays
// __global__ 变量声明符，作用是将add函数变成可以在GPU上运行的函数
// __global__ 函数被称为kernel，
// 在 GPU 上运行的代码通常称为设备代码（device code），而在 CPU 上运行的代码是主机代码（host code）。
__global__ void hi_gpu() { printf("Hello World from GPU!\n"); }

int main(int argc, char **argv) {
  XLOG_DEBUG("Hello World from CPU!");

  hi_gpu<<<1, 10>>>();
  const hipError_t err_t = hipDeviceReset();
  const std::string err_s = hipGetErrorString(err_t);
  if (err_t != hipSuccess) {
    XLOG_ERROR("CUDA error: code: {}, reason: {}", (int)err_t, err_s.c_str());
    exit(1);
  }
  return 0;
}